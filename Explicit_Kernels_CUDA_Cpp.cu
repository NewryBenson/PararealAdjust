#pragma once

#include <iostream>

//? ----------------------------------------------------------
//?
//? Description:
//?     Kernels and functions are "unified" for proper usage
//?     depending on whether GPU support is activated or not. 
//?
//? ----------------------------------------------------------
#include "Explicit.hpp"
#include "Kernels.cu"
#include "functions.hpp"
#include "Explicit_Kernels.cu"


namespace LeXInt
{
    template <typename rhs>
    void explicit_Euler(rhs RHS, double* u, double* u_sol, double* u_temp, double dt, size_t N, bool GPU)
    {
        if (GPU)
        {
            #ifdef __HIPCC__
            
            //* CUDA
            explicit_Euler_CUDA<<<(N/128) + 1, 128>>>(u, u_sol, u_temp, dt, N);

            #else
            std::cout << "Error: Compiled with GCC, not NVCC." << std::endl;
            exit(1);
            #endif
        }
        else
        {
            // CPU version
            explicit_Euler_Cpp(RHS, u, u_sol, u_temp, dt, N);
        }
    }

    template <typename rhs>
    void RK2(rhs RHS, double* u, double* u_sol, double* u_temp, double dt, size_t N, bool GPU)
    {
        if (GPU)
        {
            #ifdef __HIPCC__
            
            //* CUDA
            RK2_CUDA<<<(N/128) + 1, 128>>>(u, u_sol, u_temp, dt, N);

            #else
            std::cout << "Error: Compiled with GCC, not NVCC." << std::endl;
            exit(1);
            #endif
        }
        else
        {
            // CPU version
            RK2_Cpp(RHS, u, u_sol, u_temp, dt, N);
        }
    }

    template <typename rhs>
    void RK4(rhs RHS, double* u, double* u_sol, double* u_temp, double dt, size_t N, bool GPU)
    {
        if (GPU)
        {
            #ifdef __HIPCC__
            
            //* CUDA
            RK4_CUDA<<<(N/128) + 1, 128>>>(u, u_sol, u_temp, dt, N);

            #else
            std::cout << "Error: Compiled with GCC, not NVCC." << std::endl;
            exit(1);
            #endif
        }
        else
        {
            // CPU version
            RK4_Cpp(RHS, u, u_sol, u_temp, dt, N);
        }
    }

}