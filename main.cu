#include <iostream>
#include <hip/hip_runtime.h>

__global__ void ones_CUDA(double *x, size_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        x[idx] = 1.0;
    }
}

int main() {
    int Ntest = 1;
    double* xtest;

    //reserve shared memory
    hipMallocManaged(&xtest, Ntest * sizeof(double));

    ones_CUDA<<<(Ntest + 127) / 128, 128>>>(xtest, Ntest);

    //wait for gpu to finish
    hipDeviceSynchronize();

    if (xtest[0] == 1){std::cout << "GPU runs correctly" << std::endl;};

    //free the reserved memory
    hipFree(xtest);
    return 0;
}