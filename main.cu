#include <iostream>
#include <hip/hip_runtime.h>

__global__ void ones_CUDA(double *x, size_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        x[idx] = 1.0;
    }
}

int main() {
    int Ntest = 2;
    double* xtest;

    hipMallocManaged(&xtest, Ntest * sizeof(double));

    std::cout << "Ones says:" << std::endl;

    ones_CUDA<<<(Ntest + 127) / 128, 128>>>(xtest, Ntest);
    hipDeviceSynchronize();

    for (int i = 0; i < Ntest; ++i) {
        std::cout << xtest[i] << " ";
    }
    std::cout << std::endl;

    hipFree(xtest);
    return 0;
}