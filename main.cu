#include <iostream>
#include <hip/hip_runtime.h>

__global__ void ones_CUDA(double *x, size_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        x[idx] = 1.0;
    }
}

int main() {
    int Ntest = 2;
    double* xtest;

    hipMallocManaged(&xtest, Ntest * sizeof(double));

    std::cout << "Ones says:" << std::endl;

    ones_CUDA<<<(Ntest + 127) / 128, 128>>>(xtest, Ntest);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
    }

    // ✅ Wait for kernel to complete
    hipDeviceSynchronize();

    // ✅ Check for errors during execution
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
    }


    for (int i = 0; i < Ntest; ++i) {
        std::cout << xtest[i] << " ";
    }
    std::cout << std::endl;

    hipFree(xtest);
    return 0;
}